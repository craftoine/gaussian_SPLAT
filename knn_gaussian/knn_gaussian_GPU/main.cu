#include "hip/hip_runtime.h"
#define _USE_CUDA_
#include "../construct_tree/construct.hpp"
#include "../../common/common_header.hpp"
#include "def_functions.cuh"
#define Sort
int main(int argc, char *argv[]){
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if (deviceCount == 0)
    {
        std::cerr << "No CUDA devices found" << std::endl;
        return 1;
    }

    int device;
    for (device = 0; device < deviceCount; ++device)
    {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, device);
        if (hipGetLastError() != hipSuccess)
        {
            std::cerr << "Error getting device properties for device " << device << std::endl;
            return 1;
        }

        if (deviceProp.major >= 1)
        {
            break;
        }
    }

    if (device == deviceCount)
    {
        std::cerr << "No CUDA devices with at least compute capability 1.0 found" << std::endl;
        return 1;
    }

    hipSetDevice(device);
    if (hipGetLastError() != hipSuccess)
    {
        std::cerr << "Error setting CUDA device" << std::endl;
        return 1;
    }
    array_indexes_type N;
    std::vector<gaussian_kernel2_3D> kernels3;
    //if an input file is given the input gaussian from the input file else generate random gaussians
    if(argc>1){
        std::cout<<"Reading from file: "<<argv[1]<<std::endl;
        std::ifstream infile(argv[1],std::ios_base::binary);
        std::string buff;
        std::getline(infile,buff);
        std::getline(infile,buff);

        std::string dummy;
        std::getline(infile,buff);
        std::stringstream ss(buff);
        ss>>dummy>>dummy>>N;

        std::cout<<"N: "<<N<<std::endl;
        
        while(std::getline(infile,buff)){
            if(buff.compare("end_header") == 0)
                break;
        }

        while (infile.peek() != EOF){
                    float x, y, z;
            float nx, ny, nz;
            float opacity;
            float scale_0, scale_1, scale_2;
            float rot_0, rot_1, rot_2, rot_3;
            infile.read(reinterpret_cast<char*>(&x), sizeof(float));
            infile.read(reinterpret_cast<char*>(&y), sizeof(float));
            infile.read(reinterpret_cast<char*>(&z), sizeof(float));
            infile.read(reinterpret_cast<char*>(&nx), sizeof(float));
            infile.read(reinterpret_cast<char*>(&ny), sizeof(float));
            infile.read(reinterpret_cast<char*>(&nz), sizeof(float));
            for(int j=0;j< 3+45;j++){
                float dummy_;
                infile.read(reinterpret_cast<char*>(&dummy_), sizeof(float));
            }
            infile.read(reinterpret_cast<char*>(&opacity), sizeof(float));
            infile.read(reinterpret_cast<char*>(&scale_0), sizeof(float));
            infile.read(reinterpret_cast<char*>(&scale_1), sizeof(float));
            infile.read(reinterpret_cast<char*>(&scale_2), sizeof(float));
            infile.read(reinterpret_cast<char*>(&rot_0), sizeof(float));
            infile.read(reinterpret_cast<char*>(&rot_1), sizeof(float));
            infile.read(reinterpret_cast<char*>(&rot_2), sizeof(float));
            infile.read(reinterpret_cast<char*>(&rot_3), sizeof(float));

            //normalize the quaternion
            float norm = sqrt(rot_0*rot_0+rot_1*rot_1+rot_2*rot_2+rot_3*rot_3);
            rot_0 /= norm;
            rot_1 /= norm;
            rot_2 /= norm;
            rot_3 /= norm;

            //exponentiate the scale
            scale_0 = exp(scale_0);
            scale_1 = exp(scale_1);
            scale_2 = exp(scale_2);

            //opacity pass throught sigmoid
            opacity =( 1/(1+exp(-opacity)));

            //create scale and quaternion vectors
            float_double scale[3];
            scale[0] = (float_double) scale_0;
            scale[1] = (float_double) scale_1;
            scale[2] = (float_double) scale_2;
            float_double q[4];
            q[0] = (float_double) rot_0;
            q[1] = (float_double) rot_1;
            q[2] = (float_double) rot_2;
            q[3] = (float_double) rot_3;

            /*float_double mu[3];
            mu[0] = (float_double)x;
            mu[1] = (float_double)y;
            mu[2] = (float_double)z;*/
            point3d mu;
            mu.x = (float_double)x;
            mu.y = (float_double)y;
            mu.z = (float_double)z;

            kernels3.push_back(gaussian_kernel2_3D(mu, opacity, scale, q));
        }
    }else{
        std::cout<<"Generating random gaussians"<<std::endl;
        N = 1000000;
        for(array_indexes_type i=0;i<N;i++){
            point3d mu;
            mu.x= 1000*((float_double)rand()/(float_double)RAND_MAX);
            mu.y= 1000*((float_double)rand()/(float_double)RAND_MAX);
            mu.z= 1000*((float_double)rand()/(float_double)RAND_MAX);
            float_double* scale_ = new float_double[3];
            random_scale(scale_);
            for(dimension_type j=0;j<3;j++){
                scale_[j] = 4*scale_[j];
            }
            float_double* q = new float_double[4];
            random_unitary_quaternion(q);
            float_double weight = ((float_double)rand()/(float_double)RAND_MAX);
            kernels3.push_back(gaussian_kernel2_3D(mu, weight , scale_, q));
            free(q);
            free(scale_);
        }
    }
    std::cout<<"Number of gaussians: "<<kernels3.size()<<std::endl;
    //construct the tree
    kd_tree3 tree3(&kernels3);
    kdtree_node3 root3(&kernels3);
    tree3.root = &root3;
    std::cout<<"Tree constructed"<<std::endl;
    array_indexes_type N_points = 1000000;

    float_double min_coord3[3] = {max_float_double,max_float_double,max_float_double};
    float_double max_coord3[3] = {std::numeric_limits<float_double>::min(),std::numeric_limits<float_double>::min(),std::numeric_limits<float_double>::min()};
    min_coord3[0] = root3.range0;
    min_coord3[1] = root3.range1;
    min_coord3[2] = root3.range2;
    max_coord3[0] = root3.range3;
    max_coord3[1] = root3.range4;
    max_coord3[2] = root3.range5;

    std::vector<point3d> xs;
    for(array_indexes_type i=0;i<N_points;i++){
        point3d mu;
        mu.x= min_coord3[0] + (max_coord3[0]-min_coord3[0])*((float_double)rand()/(float_double)RAND_MAX);
        mu.y= min_coord3[1] + (max_coord3[1]-min_coord3[1])*((float_double)rand()/(float_double)RAND_MAX);
        mu.z= min_coord3[2] + (max_coord3[2]-min_coord3[2])*((float_double)rand()/(float_double)RAND_MAX);
        xs.push_back(mu);
    }
    #ifdef Sort
        std::cout<<"Sorting the points using hilbert curve"<<std::endl;
        point3d min_ = point3d(min_coord3[0],min_coord3[1],min_coord3[2]);
        point3d max_ = point3d(max_coord3[0],max_coord3[1],max_coord3[2]);
        std::vector<std::pair<point3d, double>> xs_hilbert;
        for(array_indexes_type i=0;i<N_points;i++){
            xs_hilbert.push_back(std::make_pair(xs[i], xs[i].hilbert_curve_cord(min_, max_)));
        }
        std::sort(xs_hilbert.begin(), xs_hilbert.end(), [&](std::pair<point3d, double> a, std::pair<point3d, double> b) -> bool {return a.second < b.second;});
        for(array_indexes_type i=0;i<N_points;i++){
            xs[i] = xs_hilbert[i].first;
        }
        std::cout<<"Points sorted"<<std::endl;
    #endif
    point3d* xs_ = (point3d*)malloc(N_points*sizeof(point3d));
    for(array_indexes_type i=0;i<N_points;i++){
        xs_[i] = xs[i];
    }
    char* tree3_data_GPU;
    hipMalloc(&tree3_data_GPU, root3.size);
    hipMemcpy(tree3_data_GPU, root3.data, root3.size, hipMemcpyHostToDevice);
    point3d* xs_GPU;
    hipMalloc(&xs_GPU, N_points*sizeof(point3d));
    hipMemcpy(xs_GPU, xs_, N_points*sizeof(point3d), hipMemcpyHostToDevice);
    gaussian_kernel2_3D* ress = (gaussian_kernel2_3D*)malloc(N_points*sizeof(gaussian_kernel2_3D));
    gaussian_kernel2_3D* ress_GPU;
    hipMalloc(&ress_GPU, N_points*sizeof(gaussian_kernel2_3D));
    float_double* min_dist_ = (float_double*)malloc(sizeof(float_double)*N_points);
    for(array_indexes_type i=0;i<N_points;i++){
        min_dist_[i] = max_float_double;
    }
    float_double* min_dist_GPU;
    hipMalloc(&min_dist_GPU, sizeof(float_double)*N_points);
    hipMemcpy(min_dist_GPU, min_dist_, sizeof(float_double)*N_points, hipMemcpyHostToDevice);

    gaussian_kernel2_3D* kernels = (gaussian_kernel2_3D*)malloc(N*sizeof(gaussian_kernel2_3D));
    for(array_indexes_type i=0;i<N;i++){
        kernels[i] = kernels3[i];
    }
    gaussian_kernel2_3D* kernels_GPU;
    hipMalloc(&kernels_GPU, N*sizeof(gaussian_kernel2_3D));
    hipMemcpy(kernels_GPU, kernels, N*sizeof(gaussian_kernel2_3D), hipMemcpyHostToDevice);
    int blockSize,minGridSize,gridSize;
    hipOccupancyMaxPotentialBlockSize( &minGridSize, &blockSize, 
        search1nn_arr_non_rec, 0, 0);
    int MaxGridSize;
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&MaxGridSize, search1nn_arr_non_rec, blockSize, 0);
    gridSize = std::min((size_t)((N_points + blockSize - 1) / blockSize), (size_t)MaxGridSize);
    gridSize = 82*3;
    blockSize = 128;
    std::cout << "blockSize: " << blockSize << std::endl;
    std::cout << "minGridSize: " << minGridSize << std::endl;
    std::cout << "gridSize: " << gridSize << std::endl;
    int maxActiveBlocks;
    hipOccupancyMaxActiveBlocksPerMultiprocessor(&maxActiveBlocks, search1nn_arr_non_rec, blockSize, 0);
    hipDeviceProp_t props;
    hipGetDeviceProperties(&props, device);
    float occupancy = (maxActiveBlocks * blockSize / props.warpSize) / 
                      (float)(props.maxThreadsPerMultiProcessor / 
                              props.warpSize);
    std::cout << "Occupancy: " << occupancy << std::endl;
    clock_t start, end;
//1nn search
    start = clock();
    //search1nn_arr_non_rec<<<gridSize, blockSize>>>(tree3_data_GPU, xs_GPU, ress_GPU, min_dist_GPU, kernels_GPU, N_points);
    search1nn_many_arr_non_rec_2<<<gridSize, blockSize>>>(tree3_data_GPU, xs_GPU, ress_GPU, min_dist_GPU, kernels_GPU, N_points);
    hipDeviceSynchronize();
    hipError_t error = hipGetLastError();
    if (error != hipSuccess)
    {
        std::cerr << "CUDA error: " << hipGetErrorString(error) << std::endl;
        return 1;
    }
    end = clock();
    std::cout<<"Elapsed time for 1nn : "<< (float)(end - start) / CLOCKS_PER_SEC<<std::endl;
    hipMemcpy(ress, ress_GPU, N_points*sizeof(gaussian_kernel2_3D), hipMemcpyDeviceToHost);
    hipMemcpy(min_dist_, min_dist_GPU, sizeof(float_double)*N_points, hipMemcpyDeviceToHost);

    free(ress);
    hipFree(ress_GPU);
    free(min_dist_);
    hipFree(min_dist_GPU);

//knn search
    array_indexes_type k = K_knn;
    array_indexes_type* number_found = (array_indexes_type*)malloc(sizeof(array_indexes_type)*N_points);
    for(array_indexes_type i=0;i<N_points;i++){
        number_found[i] = 0;
    }
    array_indexes_type* number_found_GPU;
    hipMalloc(&number_found_GPU, sizeof(array_indexes_type)*N_points);
    hipMemcpy(number_found_GPU, number_found, sizeof(array_indexes_type)*N_points, hipMemcpyHostToDevice);
    array_indexes_type* ress_knn = (array_indexes_type*)malloc(N_points*k*sizeof(array_indexes_type));
    array_indexes_type* ress_knn_GPU;
    hipMalloc(&ress_knn_GPU, N_points*k*sizeof(array_indexes_type));
    float_double* min_dist_knn = (float_double*)malloc(sizeof(float_double)*N_points);
    for(array_indexes_type i=0;i<N_points;i++){
        min_dist_knn[i] = max_float_double;
    }
    float_double* min_dist_knn_GPU;
    hipMalloc(&min_dist_knn_GPU, sizeof(float_double)*N_points);
    hipMemcpy(min_dist_knn_GPU, min_dist_knn, sizeof(float_double)*N_points, hipMemcpyHostToDevice);

    start = clock();
    search_knn_arr_non_rec<<<gridSize, blockSize>>>(tree3_data_GPU, xs_GPU, ress_knn_GPU, min_dist_knn_GPU, kernels_GPU, N_points, number_found_GPU);
    hipDeviceSynchronize();
    error = hipGetLastError();
    if (error != hipSuccess)
    {
        std::cerr << "CUDA error: " << hipGetErrorString(error) << std::endl;
        return 1;
    }
    end = clock();
    std::cout<<"Elapsed time for knn with k = "<<K_knn<<" : "<< (float)(end - start) / CLOCKS_PER_SEC<<std::endl;

    hipMemcpy(ress_knn, ress_knn_GPU, N_points*k*sizeof(array_indexes_type), hipMemcpyDeviceToHost);
    hipMemcpy(min_dist_knn, min_dist_knn_GPU, sizeof(float_double)*N_points, hipMemcpyDeviceToHost);
    hipMemcpy(number_found, number_found_GPU, sizeof(array_indexes_type)*N_points, hipMemcpyDeviceToHost);

    free(ress_knn);
    hipFree(ress_knn_GPU);
    free(min_dist_knn);
    hipFree(min_dist_knn_GPU);
    free(number_found);
    hipFree(number_found_GPU);

    free(kernels);
    hipFree(kernels_GPU);
    free(xs_);
    hipFree(xs_GPU);

    hipFree(tree3_data_GPU);
    return 0;
}   